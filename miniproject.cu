#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iostream>
#include <vector>
#include "cutil_inline.h"
#include <hiprand/hiprand_kernel.h>

#define GRIDSIZE 64
#define BLOCKSIZE 8
#define N 16
#define L -128.0
#define U 128.0
#define T 4096
#define TAU 64
#define R 8

using namespace std;

float host_randomUniform (float a, float b) {
    float result = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
    float min, max;
    if (a < b) {
        min = a;
        max = b;
    } else {
        min = b;
        max = a;
    }
    float diff = max - min;
    result *= diff;
    result += min;
    return result;
}

int host_randInt (int a, int b) {
    int result;
    if (a <= b) {
        result = a + rand() % (b - a);
    } else {
        result = b + rand() % (a - b);
    }
    return result;
}
/*
//Sphere function
float host_objectiveFunction (float * x) {
    float result = 0.0;
    for (int i = 0; i < N; i++) {
        result += x[i] * x[i];
    }
    return result;
}
*/
/*

float host_objectiveFunction (float * x) {
    float result = 0;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < i; j++) {
            result += x[j] * x[j];
        }
    }
    return result;
}
*/
/*
//Griewank Function
float host_objectiveFunction (float * x) {
    float result = 1;
    float sum = 0;
    float prod = 1;
    for (int i = 0; i < N; i++) {
        sum += x[i] * x[i];
        prod *= cos(x[i]/sqrt(i+1));
    }
    sum /= 4000;
    result += sum;
    result -= prod;
    return result;
}
*/
//Rastrigin Function
float host_objectiveFunction (float * x) {
    float result = 10*N;
    for (int i = 0; i < N; i++) {
        result += x[i] * x[i];
        result -= 10*cos(2*M_PI*x[i]);
    }
    return result;
}

void host_initialSolution (float * x) {
    for (int i = 0; i < N; i++) {
        x[i] = host_randomUniform(L, U);
    }
}

void host_crossover (float * h, float * x0, float * x1) {
    for (int i = 0; i < N; i++) {
        h[i] = host_randomUniform(x0[i], x1[i]);
    }
}

void host_mutation (float * y, float * x) {
    int k = host_randInt(0, N);
    for (int i = 0; i < N; i++) {
        if (i == k) {
            y[i] = host_randomUniform(L, U);
        } else {
            y[i] = x[i];
        }
    }
}

void host_findOptimum (float * solution) {
    float x0[N];
    float x1[N];
    host_initialSolution(x0);
    host_initialSolution(x1);
    for (int t = 0; t < T; t++) {
        float h[N];
        host_crossover(h, x0, x1);
        float y[N];
        host_mutation(y, h);
        if (host_objectiveFunction(x0) > host_objectiveFunction(x1)) {
            if (host_objectiveFunction(x0) > host_objectiveFunction(y)) {
                for (int i = 0; i < N; i++) {
                    x0[i] = y[i];
                }
            }
        } else {
            if (host_objectiveFunction(x1) > host_objectiveFunction(y)) {
                for (int i = 0; i < N; i++) {
                    x1[i] = y[i];
                }
            }
        }
    }
    if (host_objectiveFunction(x0) < host_objectiveFunction(x1)) {
        for (int i = 0; i < N; i++) {
            solution[i] = x0[i];
        }
    } else {
        for (int i = 0; i < N; i++) {
            solution[i] = x1[i];
        }
    }
}

__device__ float device_randomUniform (hiprandState * state, float a, float b) {
    float result = hiprand_uniform(state);
    float min, max;
    if (a < b) {
        min = a;
        max = b;
    } else {
        min = b;
        max = a;
    }
    float diff = max - min;
    result *= diff;
    result += min;
    return result;
}

__device__ int device_randInt (hiprandState * state, int a, int b) {
    int result;
    if (a <= b) {
        result = a + hiprand(state) % (b - a);
    } else {
        result = b + hiprand(state) % (a - b);
    }
    return result;
}
/*
//Sphere function
__device__ float device_objectiveFunction (float * x) {
    float result = 0.0;
    for (int i = 0; i < N; i++) {
        result += x[i] * x[i];
    }
    return result;
}
*/
/*

__device__ float device_objectiveFunction (float * x) {
    float result = 0;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < i; j++) {
            result += x[j] * x[j];
        }
    }
    return result;
}
*/
/*
//Griewank Function
__device__ float device_objectiveFunction (float * x) {
    float result = 1;
    float sum = 0;
    float prod = 1;
    for (int i = 0; i < N; i++) {
        sum += x[i] * x[i];
        prod *= cos(x[i]/sqrt(i+1));
    }
    sum /= 4000;
    result += sum;
    result -= prod;
    return result;
}
*/
//Rastrigin Function
__device__ float device_objectiveFunction (float * x) {
    float result = 10*N;
    for (int i = 0; i < N; i++) {
        result += x[i] * x[i];
        result -= 10*cos(2*M_PI*x[i]);
    }
    return result;
}

__device__ void device_initialSolution (hiprandState * state, float * x) {
    for (int i = 0; i < N; i++) {
        x[i] = device_randomUniform(state, L, U);
    }
}

__device__ void device_crossover (hiprandState * state, float * h, float * x0, float * x1) {
    for (int i = 0; i < N; i++) {
        h[i] = device_randomUniform(state, x0[i], x1[i]);
    }
}

__device__ void device_mutation (hiprandState * state, float * y, float * x) {
    int k = device_randInt(state, 0, N);
    for (int i = 0; i < N; i++) {
        if (i == k) {
            y[i] = device_randomUniform(state, L, U);
        } else {
            y[i] = x[i];
        }
    }
}

__global__ void device_findOptimum (float * solution, unsigned int seed) {
    // initialize shared mem

    hiprandState state;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state);

    float x0[N];
    float x1[N];
    device_initialSolution(&state, x0);
    device_initialSolution(&state, x1);
    for (int t = 0; t < T; t++) {
        float h[N];
        device_crossover(&state, h, x0, x1);
        float y[N];
        device_mutation(&state, y, h);
        if (device_objectiveFunction(x0) > device_objectiveFunction(x1)) {
            if (device_objectiveFunction(x0) > device_objectiveFunction(y)) {
                for (int i = 0; i < N; i++) {
                    x0[i] = y[i];
                }
            }
        } else {
            if (device_objectiveFunction(x1) > device_objectiveFunction(y)) {
                for (int i = 0; i < N; i++) {
                    x1[i] = y[i];
                }
            }
        }
        /*
            if t mod tau == 0
                write on shared mem // send the best individual -- the one with smallest objectiveFunction value
                sync threads
                read from share mem // we replace our worst individual only if the one we recieve is better
        */
    }
    if (device_objectiveFunction(x0) < device_objectiveFunction(x1)) {
        for (int i = 0; i < N; i++) {
            solution[i] = x0[i];
        }
    } else {
        for (int i = 0; i < N; i++) {
            solution[i] = x1[i];
        }
    }
    /*
        search all threads to find the best solution

        each thread put its better indvidual on shared memory
        sync threads
        thread zero search the shared memory to find the best solution 
    */
}

int main (int argc, char** argv) {
    srand (time(NULL));
    int devID;
    hipDeviceProp_t props;

    // get number of SMs on this GPU
    cutilSafeCall(hipGetDevice(&devID));
    cutilSafeCall(hipGetDeviceProperties(&props, devID));

    // allocate host memory
    unsigned int solutionMemSize = N * sizeof(float);

    float deviceSolutionValue[R];
    float deviceSolutionTime[R];

    float hostSolutionValue[R];
    float hostSolutionTime[R];

    printf("Solution size : %d\n", N);
    printf("Grid size     : %d\n", GRIDSIZE);
    printf("Block size    : %d\n", BLOCKSIZE);

    // allocate device memory
    float * deviceSolution;
    cutilSafeCall(hipMalloc((void**) &deviceSolution, solutionMemSize));

    // set up kernel for execution
    printf("Run %d Kernels.\n\n", R);

    for (int r = 0; r < R; r++) {
        printf("r == %d\n", r);
        float hostDeviceSolution[N];
        unsigned int timer = 0;
        cutilCheckError(cutCreateTimer(&timer));
        cutilCheckError(cutStartTimer(timer));

        device_findOptimum<<<GRIDSIZE, BLOCKSIZE>>>(deviceSolution, time(NULL));

        // check if kernel execution generated and error
        cutilCheckMsg("Kernel execution failed");

        // wait for device to finish
        hipDeviceSynchronize();

        // stop and destroy timer
        cutilCheckError(cutStopTimer(timer));
        deviceSolutionTime[r] = cutGetTimerValue(timer)/(1000.0);
        cutilCheckError(cutDeleteTimer(timer));

        // copy result from device to host
        cutilSafeCall(hipMemcpy(hostDeviceSolution, deviceSolution, solutionMemSize, hipMemcpyDeviceToHost));
        deviceSolutionValue[r] = host_objectiveFunction(hostDeviceSolution);
    }

    float deviceAverageSolutionValue = 0.0;
    float deviceAverageSolutionTime = 0.0;

    for (int r = 0; r < R; r++) {
        deviceAverageSolutionValue += deviceSolutionValue[r];
        deviceAverageSolutionTime += deviceSolutionTime[r];
    }

    deviceAverageSolutionValue /= R;
    deviceAverageSolutionTime /= R;

    for (int r = 0; r < R; r++) {
        clock_t begin, end;
        float hostSolution[N];
        
        begin = clock();
        host_findOptimum(hostSolution);
        end = clock();

        hostSolutionTime[r] = static_cast <float> (end - begin) / static_cast <float> (CLOCKS_PER_SEC);
        printf("tempo %f\n", hostSolutionTime[r]);
        hostSolutionValue[r] = host_objectiveFunction(hostSolution);        
    }

    float hostAverageSolutionValue = 0.0;
    float hostAverageSolutionTime = 0.0;

    for (int r = 0; r < R; r++) {
        hostAverageSolutionValue += hostSolutionValue[r];
        hostAverageSolutionTime += hostSolutionTime[r];
    }

    hostAverageSolutionValue /= R;
    hostAverageSolutionTime /= R;

    printf("Host objective function value: %f\n", hostAverageSolutionValue);
    printf("Device objective function value: %f\n", deviceAverageSolutionValue);

    // clean up memory
    cutilSafeCall(hipFree(deviceSolution));

    // exit and clean up device status
    hipDeviceReset();

    return 0;
}
