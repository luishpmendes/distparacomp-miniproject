#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iostream>
#include <vector>
#include "cutil_inline.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define GRIDSIZE 64
#define BLOCKSIZE 8
#define N 16
#define L -128.0
#define U 128.0
#define T 4096
#define TAU 64

using namespace std;

float host_randomUniform (float a, float b) {
    float result = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
    float min, max;
    if (a < b) {
        min = a;
        max = b;
    } else {
        min = b;
        max = a;
    }
    float diff = max - min;
    result *= diff;
    result += min;
    return result;
}

int host_randInt (int a, int b) {
    int result;
    if (a <= b) {
        result = a + rand() % (b - a);
    } else {
        result = b + rand() % (a - b);
    }
    return result;
}
/*
//Sphere function
float host_objectiveFunction (float * x) {
    float result = 0.0;
    for (int i = 0; i < N; i++) {
        result += x[i] * x[i];
    }
    return result;
}
*/
/*

float host_objectiveFunction (float * x) {
    float result = 0;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < i; j++) {
            result += x[j] * x[j];
        }
    }
    return result;
}
*/
/*
//Griewank Function
float host_objectiveFunction (float * x) {
    float result = 1;
    float sum = 0;
    float prod = 1;
    for (int i = 0; i < N; i++) {
        sum += x[i] * x[i];
        prod *= cos(x[i]/sqrt(i+1));
    }
    sum /= 4000;
    result += sum;
    result -= prod;
    return result;
}
*/
//Rastrigin Function
float host_objectiveFunction (float * x) {
    float result = 10*N;
    for (int i = 0; i < N; i++) {
        result += x[i] * x[i];
        result -= 10*cos(2*M_PI*x[i]);
    }
    return result;
}

void host_initialSolution (float * x) {
    for (int i = 0; i < N; i++) {
        x[i] = host_randomUniform(L, U);
    }
}
/*
void host_blxAlphaCrossover (float * h, float * x0, float * x1, float alpha) {
    for (int i = 0; i < N; i++) {
        float hMin, hMax, I;
        if (x0[i] < x1[i]) {
            hMin = x0[i];
            hMax = x1[i];
        } else {
            hMin = x1[i];
            hMax = x0[i];
        }
        I = hMax - hMin;
        h[i] = host_randomUniform(hMin - I * alpha, hMax + I * alpha);
    }
}
*/

void host_crossover (float * h, float * x0, float * x1) {
    for (int i = 0; i < N; i++) {
        h[i] = host_randomUniform(x0[i], x1[i]);
    }
}

/*
float host_delta (int t, float y, float b) {
    float result = host_randomUniform(0, 1);
    result = pow (result, static_cast <float> (1.0 - static_cast <float> (t/T)));
    result = static_cast <float> (1 - result);
    result = pow (result, b);
    result = static_cast <float> (y * result);
    return result;
}

void host_nonUniformMutation (float * y, float * x, int t, float b) {
    int k = host_randInt(0, N);
    for (int i = 0; i < N; i++) {
        if (i == k) {
            if (host_randomUniform(0, 1) >= 0.5) {
                y[i] = static_cast <float> (x[i] + host_delta(t, static_cast <float> (U - x[i]), b));
            } else {
                y[i] = static_cast <float> (x[i] - host_delta(t, static_cast <float> (x[i] - L), b));
            }
        } else {
            y[i] = x[i];
        }
    }
}
*/

void host_mutation (float * y, float * x) {
    int k = host_randInt(0, N);
    for (int i = 0; i < N; i++) {
        if (i == k) {
            y[i] = host_randomUniform(L, U);
        } else {
            y[i] = x[i];
        }
    }
}

void host_findOptimum (float * solution) {
    float x0[N];
    float x1[N];
    host_initialSolution(x0);
    host_initialSolution(x1);
    for (int t = 0; t < T; t++) {
        float h[N];
        host_crossover(h, x0, x1);
        float y[N];
        host_mutation(y, h);
        if (host_objectiveFunction(x0) > host_objectiveFunction(x1)) {
            if (host_objectiveFunction(x0) > host_objectiveFunction(y)) {
                for (int i = 0; i < N; i++) {
                    x0[i] = y[i];
                }
            }
        } else {
            if (host_objectiveFunction(x1) > host_objectiveFunction(y)) {
                for (int i = 0; i < N; i++) {
                    x1[i] = y[i];
                }
            }
        }
    }
    if (host_objectiveFunction(x0) < host_objectiveFunction(x1)) {
        for (int i = 0; i < N; i++) {
            solution[i] = x0[i];
        }
    } else {
        for (int i = 0; i < N; i++) {
            solution[i] = x1[i];
        }
    }
}

__device__ float device_randomUniform (hiprandState state, float a, float b) {
    //float result = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
    float result = hiprand_uniform(state);
    float min, max;
    if (a < b) {
        min = a;
        max = b;
    } else {
        min = b;
        max = a;
    }
    float diff = max - min;
    result *= diff;
    result += min;
    return result;
}

__device__ int device_randInt (hiprandState state, int a, int b) {
    int result;
    if (a <= b) {
        //result = a + rand() % (b - a);
        result = a + hiprand(state) % (b - a);
    } else {
        //result = b + rand() % (a - b);
        result = b + hiprand(state) % (a - b);
    }
    return result;
}
/*
//Sphere function
__device__ float device_objectiveFunction (float * x) {
    float result = 0.0;
    for (int i = 0; i < N; i++) {
        result += x[i] * x[i];
    }
    return result;
}
*/
/*

__device__ float device_objectiveFunction (float * x) {
    float result = 0;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < i; j++) {
            result += x[j] * x[j];
        }
    }
    return result;
}
*/
/*
//Griewank Function
__device__ float device_objectiveFunction (float * x) {
    float result = 1;
    float sum = 0;
    float prod = 1;
    for (int i = 0; i < N; i++) {
        sum += x[i] * x[i];
        prod *= cos(x[i]/sqrt(i+1));
    }
    sum /= 4000;
    result += sum;
    result -= prod;
    return result;
}
*/
//Rastrigin Function
__device__ float device_objectiveFunction (float * x) {
    float result = 10*N;
    for (int i = 0; i < N; i++) {
        result += x[i] * x[i];
        result -= 10*cos(2*M_PI*x[i]);
    }
    return result;
}

__device__ void device_initialSolution (float * x, hiprandState state) {
    for (int i = 0; i < N; i++) {
        x[i] = device_randomUniform(hiprandState state, L, U);
    }
}
/*
__device__ void device_blxAlphaCrossover (float * h, float * x0, float * x1, float alpha) {
    for (int i = 0; i < N; i++) {
        float hMin, hMax, I;
        if (x0[i] < x1[i]) {
            hMin = x0[i];
            hMax = x1[i];
        } else {
            hMin = x1[i];
            hMax = x0[i];
        }
        I = hMax - hMin;
        h[i] = device_randomUniform(hMin - I * alpha, hMax + I * alpha);
    }
}
*/

__device__ void device_crossover (float * h, hiprandState state, float * x0, float * x1) {
    for (int i = 0; i < N; i++) {
        h[i] = device_randomUniform(state, x0[i], x1[i]);
    }
}

/*
__device__ float device_delta (int t, float y, float b) {
    float result = device_randomUniform(0, 1);
    result = pow (result, static_cast <float> (1.0 - static_cast <float> (t/T)));
    result = static_cast <float> (1 - result);
    result = pow (result, b);
    result = static_cast <float> (y * result);
    return result;
}

__device__ void device_nonUniformMutation (float * y, float * x, int t, float b) {
    int k = device_randInt(0, N);
    for (int i = 0; i < N; i++) {
        if (i == k) {
            if (device_randomUniform(0, 1) >= 0.5) {
                y[i] = static_cast <float> (x[i] + device_delta(t, static_cast <float> (U - x[i]), b));
            } else {
                y[i] = static_cast <float> (x[i] - device_delta(t, static_cast <float> (x[i] - L), b));
            }
        } else {
            y[i] = x[i];
        }
    }
}
*/

__device__ void device_mutation (float * y, hiprandState state, float * x) {
    int k = device_randInt(0, N);
    for (int i = 0; i < N; i++) {
        if (i == k) {
            y[i] = device_randomUniform(state, L, U);
        } else {
            y[i] = x[i];
        }
    }
}

__global__ void device_findOptimum (float * solution, unsigned int seed) {
    // initialize shared mem

    hiprandState state;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state);

    float x0[N];
    float x1[N];
    device_initialSolution(x0, state);
    device_initialSolution(x1, state);
    for (int t = 0; t < T; t++) {
        float h[N];
        device_crossover(h, state, x0, x1);
        float y[N];
        device_mutation(y, state, h);
        if (device_objectiveFunction(x0) > device_objectiveFunction(x1)) {
            if (device_objectiveFunction(x0) > device_objectiveFunction(y)) {
                for (int i = 0; i < N; i++) {
                    x0[i] = y[i];
                }
            }
        } else {
            if (device_objectiveFunction(x1) > device_objectiveFunction(y)) {
                for (int i = 0; i < N; i++) {
                    x1[i] = y[i];
                }
            }
        }
        /*
            if t mod tau == 0
                write on shared mem // send the best individual -- the one with smallest objectiveFunction value
                sync threads
                read from share mem // we replace our worst individual only if the one we recieve is better
        */
    }
    if (device_objectiveFunction(x0) < device_objectiveFunction(x1)) {
        for (int i = 0; i < N; i++) {
            solution[i] = x0[i];
        }
    } else {
        for (int i = 0; i < N; i++) {
            solution[i] = x1[i];
        }
    }
    /*
        search all threads to find the best solution

        each thread put its better indvidual on shared memory
        sync threads
        thread zero search the shared memory to find the best solution 
    */
}

int main (int argc, char** argv) {
    srand (time(NULL));
    int devID;
    hipDeviceProp_t props;

    // get number of SMs on this GPU
    cutilSafeCall(hipGetDevice(&devID));
    cutilSafeCall(hipGetDeviceProperties(&props, devID));

    // allocate host memory
    unsigned int solutionMemSize = N * sizeof(float);

    float * hostSolution = (float *) malloc(solutionMemSize);

    float * hostDeviceSolution = (float *) malloc(solutionMemSize);

    printf("Solution size : %d\n", N);
    printf("Grid size     : %d\n", GRIDSIZE);
    printf("Block size    : %d\n", BLOCKSIZE);

    // allocate device memory
    float * deviceSolution;
    cutilSafeCall(hipMalloc((void**) &deviceSolution, solutionMemSize));

    // set up kernel for execution
//    printf("Run %d Kernels.\n\n", ITERS);
    unsigned int timer = 0;
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));

    device_findOptimum<<<GRIDSIZE, BLOCKSIZE>>>(deviceSolution, time(NULL));

    // loop {
    // start here
//    device_findOptimum<<<GRIDSIZE, BLOCKSIZE>>>(deviceSolution);
    // stop here
    // copy the solution found
    // copy the time delay
    // }
    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    // wait for device to finish
    hipDeviceSynchronize();

    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    double dSeconds = (cutGetTimerValue(timer)/1)/(1000.0);

    //Log througput
    printf("Time = %.4f s\n", dSeconds);
    cutilCheckError(cutDeleteTimer(timer));

    // copy result from device to host
    cutilSafeCall(hipMemcpy(hostDeviceSolution, deviceSolution, solutionMemSize, hipMemcpyDeviceToHost));

    host_findOptimum(hostSolution);

    printf("host: %f\n", host_objectiveFunction(hostSolution));
    printf("device: %f\n", host_objectiveFunction(hostDeviceSolution));

    // clean up memory
    free(hostDeviceSolution);
    free(hostSolution);
    cutilSafeCall(hipFree(deviceSolution));

    // exit and clean up device status
    hipDeviceReset();

    return 0;
}
